#include "hip/hip_runtime.h"
#include <fstream>
#include <stdio.h>
#include <iostream>

const int num_mat = 1; // total number of matrices = total number of threads const
const int N = 20; // square symmetric matrix dimension
const int nTPB = 256; // threads per block

// test symmetric matrices

double a1[N*N];

/* ---------------------------------------------------------------*/
//****************************************************************************80
//(n, a, d)
__host__ __device__
void r8mat_diag_get_vector(int n, double a[], double v[])
//****************************************************************************80
{
  int i;
  for (i = 0; i < n; i++) {
    v[i] = a[i + i * n];
  }

  return;
}
__host__ __device__
void r8mat_identity(int n, double a[]) {
  int i;
  int j;
  int k;
  k = 0;
  for (j = 0; j < n; j++) {
    for (i = 0; i < n; i++) {
      if (i == j) {
        a[k] = 1.0;
      } else {
        a[k] = 0.0;
      }
      k = k + 1;
    }
  }
  return;
}
//****************************************************************************80

//(n, a + (idx * n * n), it_max, v + (idx * n * n), d + (idx * n), it_num, rot_num)
__host__ __device__
void jacobi_eigenvalue(int n, double a[], int it_max, double v[], double d[], int &it_num, int &rot_num) {
  double * bw;
  double c;
  double g;
  double gapq;
  double h;
  int i;
  int j;
  int k;
  int l;
  int m;
  int p;
  int q;
  double s;
  double t;
  double tau;
  double term;
  double termp;
  double termq;
  double theta;
  double thresh;
  double w;
  double * zw;
  r8mat_identity(n, v);
  r8mat_diag_get_vector(n, a, d);
  bw = new double[n];
  zw = new double[n];
  for (i = 0; i < n; i++) {

    bw[i] = d[i];
    zw[i] = 0.0;
  }
  it_num = 0;
  rot_num = 0;
  while (it_num < it_max) {
    it_num = it_num + 1;
    //
    // The convergence threshold is based on the size of the elements in
    // the strict upper triangle of the matrix.
    //
    thresh = 0.0;
    for (j = 0; j < n; j++) {
      for (i = 0; i < j; i++) {
        thresh = thresh + a[i + j * n] * a[i + j * n];
      }
    }
    thresh = sqrt(thresh) / (double)(4 * n);
    if (thresh == 0.0) {
      break;
    }
    for (p = 0; p < n; p++) {
      for (q = p + 1; q < n; q++) {
        gapq = 10.0 * fabs(a[p + q * n]);
        termp = gapq + fabs(d[p]);
        termq = gapq + fabs(d[q]);
        //
        // Annihilate tiny offdiagonal elements.
        //
        if (4 < it_num &&
          termp == fabs(d[p]) && termq == fabs(d[q])) {
          a[p + q * n] = 0.0;

        }
        //
        // Otherwise, apply a rotation.
        //
        else if (thresh <= fabs(a[p + q * n])) {
          h = d[q] - d[p];
          term = fabs(h) + gapq;
          if (term == fabs(h)) {
            t = a[p + q * n] / h;
          } else {
            theta = 0.5 * h / a[p + q * n];
            t = 1.0 / (fabs(theta) + sqrt(1.0 + theta * theta));
            if (theta < 0.0) {
              t = -t;
            }
          }
          c = 1.0 / sqrt(1.0 + t * t);
          s = t * c;
          tau = s / (1.0 + c);
          h = t * a[p + q * n];
          //
          // Accumulate corrections to diagonal elements.
          //
          zw[p] = zw[p] - h;
          zw[q] = zw[q] + h;
          d[p] = d[p] - h;
          d[q] = d[q] + h;
          a[p + q * n] = 0.0;
          //
          // Rotate, using information from the upper triangle of A only.
          //
          for (j = 0; j < p; j++) {
            g = a[j + p * n];
            h = a[j + q * n];
            a[j + p * n] = g - s * (h + g * tau);
            a[j + q * n] = h + s * (g - h * tau);
          }

          for (j = p + 1; j < q; j++) {
            g = a[p + j * n];
            h = a[j + q * n];
            a[p + j * n] = g - s * (h + g * tau);
            a[j + q * n] = h + s * (g - h * tau);
          }
          for (j = q + 1; j < n; j++) {
            g = a[p + j * n];
            h = a[q + j * n];
            a[p + j * n] = g - s * (h + g * tau);
            a[q + j * n] = h + s * (g - h * tau);
          }
          //
          // Accumulate information in the eigenvector matrix.
          //
          for (j = 0; j < n; j++) {
            g = v[j + p * n];
            h = v[j + q * n];
            v[j + p * n] = g - s * (h + g * tau);
            v[j + q * n] = h + s * (g - h * tau);
          }
          rot_num = rot_num + 1;
        }
      }
    }
    for (i = 0; i < n; i++) {
      bw[i] = bw[i] + zw[i];
      d[i] = bw[i];
      zw[i] = 0.0;
    }
  }
  //
  // Restore upper triangle of input matrix.
  //
  for (j = 0; j < n; j++) {
    for (i = 0; i < j; i++) {
      a[i + j * n] = a[j + i * n];
    }
  }

  //
  // Ascending sort the eigenvalues and eigenvectors.
  //
  for (k = 0; k < n - 1; k++) {
    m = k;
    for (l = k + 1; l < n; l++) {
      if (d[l] < d[m]) {
        m = l;
      }
    }
    if (m != k) {
      t = d[m];
      d[m] = d[k];
      d[k] = t;
      for (i = 0; i < n; i++) {
        w = v[i + m * n];
        v[i + m * n] = v[i + k * n];
        v[i + k * n] = w;
      }
    }
  }
  delete[] bw;
  delete[] zw;
  return;
}

// end of FSU code
/* ---------------------------------------------------------------- */

//(num_mat, N,d_a, max_iter, d_v, d_d)
__global__ void je(int num_matr, int n, double *a, int it_max, double *v, double *d) {
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  int it_num;
  int rot_num;
  if (idx < num_matr) {
    jacobi_eigenvalue(n, a + (idx * n * n), it_max, v + (idx * n * n), d + (idx * n), it_num, rot_num);
  }
}

//(0, N, a1, h_a)
void initialize_matrix(int mat_id, int n, double * mat, double * v) {
  for (int i = 0; i < n * n; i++) * (v + (mat_id * n * n) + i) = mat[i];
}
void print_vec(int vec_id, int n, double * d) {
  std::cout << "matrix " << vec_id << " Diagonal Values: " << std::endl;
  std::cout.precision(12);
  for (int i = 0; i < n; i++) std::cout << i << ": " << * (d + (n * vec_id) + i) << std::endl;
  std::cout << std::endl;
}
int main() {
  // make sure device heap has enough space for in-kernel new allocations
  const int heapsize = num_mat * N * sizeof(double) * 2;
  const int chunks = heapsize / (8192 * 1024) + 1;
  hipError_t cudaStatus = hipDeviceSetLimit(hipLimitMallocHeapSize, (8192 * 1024) * chunks);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "set device heap limit failed!");
  }


  //reading matrix from file matrix.dat
  std::ifstream in;
  in.open("data/matrix.dat");
	if (!in)
	{
		std::cerr << "Unable to open file containing matrix. It should have the name matrix.dat. It should be present inside the data folder" << std::endl;
		exit(1);
	}
	else
	{
		for (int i = 0; i < N*N; i++)
		{
			in >> a1[i];
		}
 	  in.close();
  }

  const int max_iter = 1000;
  double *h_a, *d_a, *h_v, *d_v, *h_d, *d_d;

  //initialize_matrix
  h_a = (double * ) malloc(num_mat * N * N * sizeof(double));
  h_v = (double *) malloc(num_mat * N * N * sizeof(double));
  h_d = (double * ) malloc(num_mat * N * sizeof(double));

  //Allocating mem in GPU
  hipMalloc( & d_a, num_mat * N * N * sizeof(double));
  hipMalloc( & d_v, num_mat * N * N * sizeof(double));
  hipMalloc( & d_d, num_mat * N * sizeof(double));

  //sets all the bytes to a specific value (0)
  memset(h_a, 0, num_mat * N * N * sizeof(double));
  memset(h_v, 0, num_mat * N * N * sizeof(double));
  memset(h_d, 0, num_mat * N * sizeof(double));

  for(int i = 0; i< num_mat ; i++)
  initialize_matrix(i, N, a1, h_a);

  //initialize_matrix(1, N, a2, h_a);
  //initialize_matrix(2, N, a3, h_a);

  //copys variable to GPU
  hipMemcpy(d_a, h_a, num_mat * N * N * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_v,h_v, num_mat * N * N * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_d, h_d, num_mat *N * sizeof(double), hipMemcpyHostToDevice);


  clock_t cpu_startTime, cpu_endTime;
  double cpu_ElapseTime=0;
  cpu_startTime = clock();

  //Launches Kernel
  //<<<N/BLOCK_SIZE,BLOCK_SIZE>>>
  je << < (num_mat + nTPB - 1) / nTPB, nTPB >>> (num_mat, N,d_a, max_iter, d_v, d_d);

  cpu_endTime = clock();
  cpu_ElapseTime = ((cpu_endTime - cpu_startTime)/(double)CLOCKS_PER_SEC);
  std::cout << "CPU Elapsed time: " <<cpu_ElapseTime<<"s\n";


  //Copy back to CPU
  hipMemcpy(h_d, d_d, num_mat * N * sizeof(double), hipMemcpyDeviceToHost);


  //for(int i = 0; i< num_mat ; i++)
  //  print_vec(i, N, h_d);

  print_vec(0, N, h_d);
  //print_vec(2, N, h_d);

  return 0;
}
